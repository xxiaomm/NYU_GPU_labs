#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define BLOCK_NUM 8
#define BLOCK_SIZE 500

#define RANGE 19.87

/*** Declaration of the kernel function below this line ***/

__global__ void vecGPU(float *a, float *b, float *c, int n, int stride);

/**** end of the kernel declaration ***/

int main(int argc, char *argv[])
{

	int n = 0;					 // number of elements in the arrays
	int i;							 // loop index
	float *a, *b, *c;		 // The arrays that will be processed in the host.
	float *temp;				 // array in host used in the sequential code.
	float *ad, *bd, *cd; // The arrays that will be processed in the device.
	clock_t start, end;	 // to meaure the time taken by a specific part of code

	if (argc != 2)
	{
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
	}

	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);

	// Allocating the arrays in the host

	if (!(a = (float *)malloc(n * sizeof(float))))
	{
		printf("Error allocating array a\n");
		exit(1);
	}

	if (!(b = (float *)malloc(n * sizeof(float))))
	{
		printf("Error allocating array b\n");
		exit(1);
	}

	if (!(c = (float *)malloc(n * sizeof(float))))
	{
		printf("Error allocating array c\n");
		exit(1);
	}

	if (!(temp = (float *)malloc(n * sizeof(float))))
	{
		printf("Error allocating array temp\n");
		exit(1);
	}

	// Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n; i++)
	{
		a[i] = ((float)rand() / (float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand() / (float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand() / (float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; // temp is just another copy of C
	}

	// The sequential part
	start = clock();
	for (i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

	/******************  The start GPU part: Do not modify anything in main() above this line  ************/
	// The GPU part

	// Allocate the arrays in the device
	hipMalloc((void **)&ad, n * sizeof(float));
	hipMalloc((void **)&bd, n * sizeof(float));
	hipMalloc((void **)&cd, n * sizeof(float));

	// Copy the arrays from the host to the device
	hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bd, b, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cd, c, n * sizeof(float), hipMemcpyHostToDevice);

	start = clock();

	// Call the kernel function
	dim3 grid_size(BLOCK_NUM, 1, 1);
	dim3 block_size(BLOCK_SIZE, 1, 1);
	int stride = ceil(n / (float)(grid_size.x * block_size.x));
	vecGPU<<<grid_size, block_size>>>(ad, bd, cd, n, stride);

	// Force host to wait on the completion of the kernel
	hipDeviceSynchronize();

	end = clock();

	// Copy the result from the device to the host
	hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost);

	// Free the memory allocated in the device
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);

	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/

	// Checking the correctness of the GPU part
	for (i = 0; i < n; i++)
		if (fabs(temp[i] - c[i]) >= 0.009) // compare up to the second degit in floating point
			printf("Element %d in the result array does not match the sequential version\n", i);

	// Free the arrays in the host
	free(a);
	free(b);
	free(c);
	free(temp);

	return 0;
}

/**** Write the kernel itself below this line *****/

__global__ void vecGPU(float *a, float *b, float *c, int n, int stride)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int j = i * stride; j < (i + 1) * stride && j < n; j++)
	{
		c[j] += a[j] * b[j];
	}
}
